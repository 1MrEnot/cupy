#include "cupy_cufftx.h"


// this must define d_loadCallbackPtr
${dev_load_callback_ker}

// this must define d_storeCallbackPtr
${dev_store_callback_ker}

hipfftResult set_callback(hipfftHandle plan, hipfftXtCallbackType type, bool cb_load) {
    if (cb_load) {  // for load callback
        switch (type) {
            case HIPFFT_CB_LD_COMPLEX: {
                hipfftCallbackLoadC h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            case HIPFFT_CB_LD_COMPLEX_DOUBLE: {
                hipfftCallbackLoadZ h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            case HIPFFT_CB_LD_REAL: {
                hipfftCallbackLoadR h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            case HIPFFT_CB_LD_REAL_DOUBLE: {
                hipfftCallbackLoadD h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            default: {
                throw std::runtime_error("unrecognized callback");
            }
        }
    } else {  // for store callback
        switch (type) {
            case HIPFFT_CB_ST_COMPLEX: {
                hipfftCallbackStoreC h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            case HIPFFT_CB_ST_COMPLEX_DOUBLE: {
                hipfftCallbackStoreZ h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            case HIPFFT_CB_ST_REAL: {
                hipfftCallbackStoreR h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            case HIPFFT_CB_ST_REAL_DOUBLE: {
                hipfftCallbackStoreD h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            default: {
                throw std::runtime_error("unrecognized callback");
            }
        }
    }
}
