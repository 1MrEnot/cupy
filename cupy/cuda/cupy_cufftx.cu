#include "hip/hip_runtime.h"
#include "cupy_cufftx.h"


// this defines d_loadCallbackPtr
${dev_load_callback_ker}

// this defines  d_storeCallbackPtr
${dev_store_callback_ker}

//
///// Device-globals to keep function pointers
///// These need to be set on the device, copied to host,
///// and then passed to the cuFFT plan.
//
//__device__ hipfftCallbackLoadC d_loadCallbackCPtr; 
////__device__ hipfftCallbackLoadZ d_loadCallbackZPtr; 
////__device__ hipfftCallbackLoadR d_loadCallbackRPtr; 
////__device__ hipfftCallbackLoadD d_loadCallbackDPtr; 
//__device__ hipfftCallbackStoreC d_storeCallbackCPtr;
////__device__ hipfftCallbackStoreZ d_storeCallbackZPtr;
////__device__ hipfftCallbackStoreR d_storeCallbackRPtr;
////__device__ hipfftCallbackStoreD d_storeCallbackDPtr;
//
//
//__global__ void setLoadCallbackCPtr(intptr_t dev_ptr) {
//    d_loadCallbackCPtr = (hipfftCallbackLoadC)dev_ptr;
//}
//
//__global__ void setStoreCallbackCPtr(intptr_t dev_ptr) {
//    d_storeCallbackCPtr = (hipfftCallbackStoreC)dev_ptr;
//}


hipfftResult set_callback(hipfftHandle plan, hipfftXtCallbackType type, bool cb_load) {
    if (cb_load) {  // for load callback
        switch (type) {
            case HIPFFT_CB_LD_COMPLEX: {
                hipfftCallbackLoadC h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, NULL);
            }
            default: {
                throw std::runtime_error("unrecognized callback");
            }
        }
    } else {  // for store callback
        switch (type) {
            default: {
                throw std::runtime_error("unrecognized callback");
            }
        }
    }
}
