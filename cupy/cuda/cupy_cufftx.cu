#include "hip/hip_runtime.h"
#include "cupy_cufft.h"
#include <cstdint>


/// Device-globals to keep function pointers
/// These need to be set on the device, copied to host,
/// and then passed to the cuFFT plan.

__device__ hipfftCallbackLoadC d_loadCallbackCPtr; 
//__device__ hipfftCallbackLoadZ d_loadCallbackZPtr; 
//__device__ hipfftCallbackLoadR d_loadCallbackRPtr; 
//__device__ hipfftCallbackLoadD d_loadCallbackDPtr; 
__device__ hipfftCallbackStoreC d_storeCallbackCPtr;
//__device__ hipfftCallbackStoreZ d_storeCallbackZPtr;
//__device__ hipfftCallbackStoreR d_storeCallbackRPtr;
//__device__ hipfftCallbackStoreD d_storeCallbackDPtr;


__global__ void setLoadCallbackCPtr(intptr_t dev_ptr) {
    d_loadCallbackCPtr = (hipfftCallbackLoadC)dev_ptr;
}

__global__ void setStoreCallbackCPtr(intptr_t dev_ptr) {
    d_storeCallbackCPtr = (hipfftCallbackStoreC)dev_ptr;
}


//hipfftResult inline setCallback(hipfftHandle plan, void **callbackRoutine, hipfftXtCallbackType type, void **callerInfo) {
//    return hipfftXtSetCallback(plan, callbackRoutine, type, callerInfo);
//}
